#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void helloKernel() {
    printf("Hello from thread %d of block %d\n!", threadIdx.x, blockIdx.x);
}

int main() {
    printf("Hello from the CPU\n");
    helloKernel <<<2, 4>>> ();
    hipDeviceSynchronize();

    hipError_t error = hipGetLastError();
    if(error != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(error));
        exit(-1);
    };
    return 0;
}
