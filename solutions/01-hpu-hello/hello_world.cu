#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void helloKernel ()
{
  printf ("Hello from the GPU!\n");
}

int
main ()
{
  printf ("Hello from the CPU\n");
  helloKernel <<< 1, 1 >>> ();
  hipDeviceSynchronize ();
  return 0;
}
