#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void helloKernel() {
    printf("Hello from thread %d of block %d\n!", threadIdx.x, blockIdx.x);
}

int main() {
    printf("Hello from the CPU\n");
    helloKernel <<<1, 1024>>> ();
    hipDeviceSynchronize();

    hipError_t error = hipGetLastError();
    if(error != hipSuccess) {
        // print the CUDA error message and exit
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    };
    return 0;
}
