#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void add(int *a, int *b, int *c)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
	c[index] = a[index] + b[index];
}

#define N (2048*2048)
#define THREADS_PER_BLOCK 512

int main()
{
    int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int size = N * sizeof( int );

	/* allocate space for host copies of a, b, c and setup input values */

	a = (int *)malloc( size );
	b = (int *)malloc( size );
	c = (int *)malloc( size );

	/* allocate space for device copies of a, b, c */
    /* ----- YOUR CODE HERE ----- */




    /* -------------------------- */

	for( int i = 0; i < N; i++ )
	{
		a[i] = b[i] = i;
		c[i] = 0;
	}

	/* copy inputs to device */
    /* ----- YOUR CODE HERE ----- */




    /* -------------------------- */


	/* launch the kernel on the GPU */

	add<<< N / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>( d_a, d_b, d_c );

	/* copy result back to host */

	hipMemcpy( c, d_c, size, hipMemcpyDeviceToHost );

	printf( "c[0] = %d\n",0,c[0] );
	printf( "c[%d] = %d\n",N-1, c[N-1] );

	/* clean up */

	free(a);
	free(b);
	free(c);
	hipFree( d_a );
	hipFree( d_b );
	hipFree( d_c );
	
	return 0;
} /* end main */

